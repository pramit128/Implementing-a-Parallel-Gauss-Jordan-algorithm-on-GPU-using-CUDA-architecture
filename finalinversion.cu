#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include "rowExchange.h"
#include "fixRows.h"
#include "fixColumns.h"
#define seed 1
#include "inputmatrix.h"

	
int main()
{
	FILE *outFile, *inFile, *vFile;
	
	inFile = fopen("matrix.txt","w");
	
	printf("Opening file matrix.txt...\n\n");
	//check if there is error in opening file
	
	if(inFile==NULL)
	{
		perror("File read error.");
		exit(1);
	}

	outFile = fopen("inverse.txt","w");
	
	printf("Opening file inverse.txt...\n\n");
	
	//check if there is error in opening file
	if(outFile==NULL)
	{
		perror("File read error.");
		exit(1);
	}
	
	int size, columns, threads;
	int choice;
	int j = 0;
	
	printf("Please enter the size of input matrix. Enter x for matrix of size x*x\n");
	scanf("%d", &size);
	
	columns = 2*size; //double the size of the matrix for augmenting the identity matrix
		
	if(columns < 1025)
		threads = columns % 1025;	
	else
		threads = 1024;
		
	dim3 thread(threads);
	dim3 rBlock((int)ceil(columns/threads)+1);
	dim3 cBlock((int)ceil(size*columns/threads)+1);
	
	float *matrix, *inMatrix, *rMatrix, *invMatrix;		
	float *devMatrix;	//the device matrix
	
	printf("\nAllocating memory in CPU...\n");
	matrix = (float*)malloc(size*columns*sizeof(float));	//allocate memory
	inMatrix = (float*)malloc(size*size*sizeof(float));	//allocate memory
	rMatrix = (float*)malloc(size*size*sizeof(float));	//allocate memory
	invMatrix = (float*)malloc(size*size*sizeof(float));	//allocate memory
	
	printf("\nAllocating memory in CPU complete...\n");

	printf("\nEnter the type of matrix you wish to generate\n");
	printf("1 Random Matrix\n");
	printf("2 Sparse Matrix\n");
	printf("3 Identity Matrix\n");
	printf("4 Band Matrix\n");
	printf("5 Hollow Matrix\n");
	printf("6 Exit\n");
	scanf("%d", &choice);

	switch(choice)
	{
		case 1:
			randomMatrix(matrix, size, columns);
			printf("\nRandom matrix generated...\n");
			break;
							
		case 2:
			sparseMatrix(matrix, size, columns);
			printf("\nSparse Matrix generated...\n");
			break;
							
		case 3:
			identityMatrix(matrix, size, columns);
			printf("\nIdentity Matrix generated...\n");
			break;
							
		case 4:
			bandMatrix(matrix, size, columns);
			printf("\nBand Matrix generated...\n");
			break;

		case 5:
			hollowMatrix(matrix, size, columns);
			printf("\nHollow Matrix generated...\n");
			break;
							
		case 6: 
			printf("\nExiting program...\n");
			exit(0);
			break;
		default:
			printf("\nInvalid Entry");
			printf("Exiting program...\n\n");
			exit(0);
			break;
				
				
	}
	
	printf("\nWriting input matrix to matrix.txt...\n");

	//print initialized matrix[] in a text file
	fprintf(inFile,":::::INPUT MATRIX IS:::::\n\n");
	for(int i=0; i<size; i++)
	{
		for(int j=0; j<size; j++)
		{	
			//Initialize matrices to verify inverse computation
			inMatrix[i*size+j] = matrix[i*columns+j];
			rMatrix[i*size+j] = 0;
			fprintf(inFile,"%.3f\t", matrix[i*columns+j]);
		}
		fprintf(inFile,"\n\n");
	}

	printf("\nWriting input matrix to matrix.txt complete...\n");
	
	//Declare event object variables
	hipEvent_t start, finish;
	
	//Time taken for computation
	float duration;
	
	//Create event objects
	hipEventCreate(&start);
	hipEventCreate(&finish);

	//allocate memory in the device
	hipMalloc((void**)&devMatrix, size * columns * sizeof(float));

	//Record event at the before start of computation
	hipEventRecord(start, 0);
	
	printf("\nComputing Inverse...\n");

	//Wait until event start is actually recorded.
	hipEventSynchronize(start);

	//copy matrix from host memory to device memory
	hipMemcpy(devMatrix, matrix, size * columns * sizeof(float), hipMemcpyHostToDevice);
	
	
	while(j<size)
	{
		if(matrix[j*columns+j]==0)
		{
			int i = 0;
			for(i=j+1; i<size; i++)
			{
				if(matrix[i*columns+j] != 0)
					break;
			}

			//Call kernel rowExchange() that makes diagonal element non zero
			rowExchange<<<rBlock, thread>>>(devMatrix, columns, j, i);
		}
		
		//Make diagonal element 1 by dividing whole row by itself
		fixRows<<<rBlock, thread>>>(devMatrix, columns, j);

		//Make elements in a column zero except diagonal element
		fixColumns<<<cBlock, thread>>>(devMatrix, columns, j);
		
		j++;
	}
	
	//copy matrix from device memory to host memory
	hipMemcpy(matrix, devMatrix, size * columns * sizeof(float), hipMemcpyDeviceToHost);

	//Record event after the end of computation
	hipEventRecord(finish, 0);
	
	printf("\nInverse for the input matrix has been computed...\n");
	
	//Wait until event finish is actually recorded.
	hipEventSynchronize(finish);
	
	//The time elapsed (in milliseconds) between events start and finish will be stored in variable duration.
	hipEventElapsedTime(&duration, start, finish);

	//Destroy start event
	hipEventDestroy(start);
	
	//Destroy finish event
	hipEventDestroy(finish);

	//Free device memory
	hipFree(devMatrix);
	
	printf("\nThe time taken for computing the inverse of the input matrix is = %.3f MILLISECONDS\n",duration);
	
	printf("\nWriting to inverse.txt...\n");

	//print inverse of given matrix computed by applying Gauss Jordan Elimination
	fprintf(outFile,":::::INVERSE OF GIVEN MATRIX IS:::::\n\n");
	for(int i=0; i<size; i++)
	{
		
		for(int j=size; j<columns; j++)
		{
			invMatrix[i*size+j-size] = matrix[i*columns+j];
			fprintf(outFile,"%.3f\t", matrix[i*columns+j]);
		}
		fprintf(outFile,"\n\n");
	}
	
	printf("\nWriting to inverse.txt complete...\n");

	//print computation time on console and output file
	
	fprintf(outFile,":::TIME TAKEN FOR COMPUTATION IS::::\t%.3f MILLISECONDS\n\n",duration);
	
	printf("\nClosing file inverse.txt...\n");
	fclose(outFile);
	printf("\nClosing file matrix.txt...\n");
	fclose(inFile);

	/*
	
	printf("\nPreparing for verification by multiplying the input matrix and the computed inverse matrix...\n");
	
	//Multiply input matrix and resulting inverse matrix so that it can result into identity matrix
	//If the result is identity matrix, the computation is justified to be correct
	for(int i=0; i<size; i++)
		for(int j=0; j<size; j++)
			for(int k=0; k<size; k++)
				rMatrix[i*size+j] += inMatrix[i*size+k]*invMatrix[k*size+j];
	//End of matrix multiplication logic
	
	printf("\nOpening file verify.txt...\n");
	
	vFile = fopen("verify.txt","w");
	//check if there is error in opening file
	if(vFile==NULL)
	{
		perror("File read error.");
		exit(1);
	}
	
	printf("\nWriting to file verify.txt...\n");
	
	fprintf(vFile,"\n :::Multiplication of input matrix and inverse matrix is::: \n");
	
	printf("\nWriting to file verify.txt complete...\n");
	
	for(int i=0; i<size; i++)
	{
		
		for(int j=0; j<size; j++)
		{
			fprintf(vFile, "%1.0f\t", rMatrix[i*size+j]);
		}
		fprintf(vFile, "\n\n");
	}
	
	printf("\nClosing file verify.txt...\n");
	fclose(vFile);
	*/
	printf("\nFree up memory...\n");
	free(matrix);
	free(inMatrix);
	free(rMatrix);
	free(invMatrix);
	printf("\nProgram complete...\n");
	return 0;
}

